#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <SDL.h>

#include "rtweekend.h"

#include "hittable.cuh"
#include "hittable_list.h"
#include "sphere.cuh"
#include "camera.cuh"
#include "material.cuh"

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, int numXPixels, int numYPixels) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(-1,0,-1), -0.4,
                               new dielectric(1.5));
        d_list[1] = new sphere(vec3(0,0,-1), 0.5,
                               new lambertian(color(0.8, 0.3, 0.3)));
        d_list[2] = new sphere(vec3(1,0,-1), 0.5,
                               new metal(color(0.8, 0.6, 0.2), 0.1));
        d_list[3] = new sphere(vec3(0,-100.5,-1), 100,
                               new lambertian(color(0.8, 0.8, 0.0))); // Ground
        *d_world  = new hittable_list(d_list, 4);
        *d_camera = new camera(point3(3,3,2), point3(0,0,-1), vec3(0,1,0),
                               20.0, float(numXPixels) / float(numYPixels));
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera) {
    for(int i=0; i < 4; i++) {
        delete ((sphere *)d_list[i])->mat;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void update_camera_position(bool w, bool a, bool s, bool d, camera **d_camera) {
    vec3 forwardDirection = normalize((*d_camera)->target - (*d_camera)->origin);
    vec3 rightDirection = normalize(cross(forwardDirection, (*d_camera)->view_up));
    if (w) {
        (*d_camera)->origin += forwardDirection * -moveSpeed;
        (*d_camera)->target += forwardDirection * -moveSpeed;
    } else if (s) {
        (*d_camera)->origin += forwardDirection * moveSpeed;
        (*d_camera)->target += forwardDirection * moveSpeed;
    } else if (a) {
        (*d_camera)->origin += rightDirection * moveSpeed;
        (*d_camera)->target += rightDirection * moveSpeed;
    } else if (d) {
        (*d_camera)->origin += rightDirection * -moveSpeed;
        (*d_camera)->target += rightDirection * -moveSpeed;
    }
}

int main() {
    std::ofstream fout;
    fout.open("image.ppm");

    if (!fout.is_open()) {
        std::cerr << "Error opening file!" << std::endl;
        return 1;
    }

    int numXPixels = 1200;
    int numYPixels = 600;
    int numSamples = 50;
    int tilesX = 8;
    int tilesY = 8;

    std::cerr << "Rendering a " << numXPixels << "x" << numYPixels << " image with " << numSamples << " samples per pixel ";
    std::cerr << "in " << tilesX << "x" << tilesY << " blocks.\n";

    int num_pixels = numXPixels * numYPixels;
    size_t fb_size = num_pixels*sizeof(vec3);

    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 4*sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, numXPixels, numYPixels);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(numXPixels / tilesX + 1, numYPixels / tilesY + 1);
    dim3 threads(tilesX, tilesY);
    render_init<<<blocks, threads>>>(numXPixels, numYPixels, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // We render a super high quality one so we can save the image
    render<<<blocks, threads>>>(fb, numXPixels, numYPixels, 1000, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    fout << "P3\n" << numXPixels << " " << numYPixels << "\n255\n";
    for (int j = numYPixels - 1; j >= 0; j--) {
        for (int i = 0; i < numXPixels; i++) {
            size_t pixel_index = j * numXPixels + i;
            write_color(fout, fb[pixel_index]);
        }
    }

    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "SDL could not initialize! SDL_Error: " << SDL_GetError() << std::endl;
        return -1;
    }

    SDL_Window* window = SDL_CreateWindow("CUDA Ray Tracing", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, numXPixels, numYPixels, SDL_WINDOW_SHOWN | SDL_WINDOW_OPENGL);
    if (!window) {
        std::cerr << "Window could not be created! SDL_Error: " << SDL_GetError() << std::endl;
        return -1;
    }

    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_ARGB8888, SDL_TEXTUREACCESS_STREAMING, numXPixels, numYPixels);

    SDL_WarpMouseInWindow(window, numXPixels / 2, numYPixels / 2);

    SDL_ShowCursor(SDL_DISABLE);
    bool quit = false;
    SDL_Event e;
    int mouseXDelta, mouseYDelta;

    while (!quit) {
        while (SDL_PollEvent(&e) != 0) {
            if (e.type == SDL_QUIT) {
                quit = true;
            }

            if (e.type == SDL_KEYDOWN) {
                switch (e.key.keysym.sym) {
                    case SDLK_w:
                        update_camera_position<<<1, 1>>>(true, false, false, false, d_camera);
                        break;
                    case SDLK_s:
                        update_camera_position<<<1, 1>>>(false, false, true, false, d_camera);
                        break;
                    case SDLK_a:
                        update_camera_position<<<1, 1>>>(false, true, false, false, d_camera);
                        break;
                    case SDLK_d:
                        update_camera_position<<<1, 1>>>(false, false, false, true, d_camera);
                        break;
                }
            }
        }

        SDL_GetRelativeMouseState(&mouseXDelta, &mouseYDelta);

        updateTextureFromFrameBuffer(texture, fb, numXPixels, numYPixels);

        SDL_RenderClear(renderer);
        SDL_RenderCopy(renderer, texture, NULL, NULL);

        SDL_RenderPresent(renderer);

        render<<<blocks, threads>>>(fb, numXPixels, numYPixels, numSamples, d_camera, d_world, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    }

    std::cout << "Cleaning up" << std::endl;

    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();

    fout.close();

    return 0;
}
